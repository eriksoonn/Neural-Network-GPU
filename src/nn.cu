#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h> 
#include <math.h>
#include <time.h>
#include "ds.h"
#include "nn.cuh"
#include "nn_aux.cuh"
#include "utils.h"
#include "matrix.cuh"
#include "test.h"
#include "train.h"
#include "globals.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <omp.h>

void init_nn(nn_t *nn, int n_layers, int *layers_size){
    int i;

    nn->n_layers = n_layers;
    nn->layers_size = layers_size;
    nn->init_weight_ptr = init_weight_rnd;
    nn->activation_ptr= (activation_ptr_t*)malloc((nn->n_layers - 1) * sizeof(activation_ptr_t));
    nn->dactivation_ptr= (activation_ptr_t*)malloc((nn->n_layers - 1) * sizeof(activation_ptr_t));
    for(i = 0; i < n_layers - 1; i++){
        nn->activation_ptr[i] = sigmoid;
        nn->dactivation_ptr[i] = dSigmoid;
    }
    nn->loss = mse;
    nn->BH = alloc_matrix_1v(n_layers - 1, &layers_size[1], nn->init_weight_ptr);
    nn->WH = alloc_matrix_2v(n_layers - 1, &layers_size[1], &layers_size[0], nn->init_weight_ptr);
}

//A[batch_id][sample_in_batch][][]
__global__ void batch_forward_pass(nn_t *nn, ds_t *ds, double *A, double *Z, int batch_size, int batch_number, int *order, int matrix_size) {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    int min_batch, i;

    //shulle order one thread

    if (threadId < batch_number) {
        for (int batch_i = 0; batch_i < batch_size; batch_i++) {
            min_batch = batch_i * batch_size;
            i = order[min_batch];

            //matrix indexing inside tensor
            unsigned int index = threadId * (batch_size * matrix_size) + batch_i * matrix_size;

            //printf("good %d = %d * ( %d * %d ) + %d * %d\n", index, threadId, batch_size, matrix_size, batch_i, matrix_size);
            forward_pass_kernel(nn, &ds->inputs[i * ds->n_inputs], &A[index], &Z[index], batch_i);

/*             if (threadId == 19900) {
                printf("%d = %d * ( %d * %d ) + %d * %d\n", index, threadId, batch_size, matrix_size, batch_i, matrix_size);

                printf("Z ");
                int j3 = index_counter(nn->layers_size, 2 - 1);
                for (int i = 0; i < 10; i++) {
                    printf("%f ", Z[index + j3 + i]);
                }
                printf("\n");
            } */
        }      
    }
}

__host__ __device__ int index_counter(int *sizes, int index) {
    int counter = 0;

    if (index == 0) {
        return counter;
    }

    for (int i = 0; i < index; i++) {
        counter +=  sizes[i];
    }

    return counter;
}

__device__ void forward_pass_kernel(nn_t *nn, double *input, double *A, double *Z, int sample_in_batch) {
    int j, j2;

    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    for(int i = 0; i < nn->layers_size[0]; i++){        
        A[i] = input[i];
    }

/*     if (threadId == 0) {
        printf("A inside ");
        for (int i = 0; i < 10; i++) {
            printf("%f ", A[i]);
        }
        printf("\n");
    } */
    
    for(int i = 1; i < nn->n_layers; i++){
        //INDEX LEVEL HACKING!!
        j = index_counter(nn->layers_size, i);
        j2 = index_counter(nn->layers_size, i - 1);
        matrix_mul_add(&Z[j], nn->WH[i - 1], &A[j2],  nn->layers_size[i], nn->layers_size[i - 1], nn->layers_size[i - 1], 1, nn->BH[i - 1]);  
        matrix_func(&A[j], &Z[j], nn->layers_size[i], 1, sigmoid);
        matrix_func(&Z[j], &Z[j], nn->layers_size[i], 1, dSigmoid);
    }

/*     if (threadId == 0) {
        printf("Z inside ");
        int j3 = index_counter(nn->layers_size, 2 - 1);
        for (int i = 0; i < 10; i++) {
            printf("%f ", Z[j3 + i]);
        }
        printf("\n");
    } */
}

__global__ void batch_back_prop(nn_t *nn, double *output, double *A, double *Z, double *D, double *d, double *E, double *D_aux, int batch_size, int batch_number, int *order, int matrix_size) {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    int min_batch, i;

    if (threadId < batch_number) {
        for (int batch_i = 0; batch_i < batch_size; batch_i++) {
            min_batch = batch_i * batch_size;
            i = order[min_batch];

            //matrix indexing inside tensor
            unsigned int index = threadId * (batch_size * matrix_size) + batch_i * matrix_size;
            unsigned int index_2 = threadId * matrix_size;

            //back_prop_kernel(nn, output, &A[index], &Z[index], &D[index_2], &d[index_2], &E[index_2], &D_aux[index_2]);
        }      
    }
}

/* __device__ void back_prop_kernel(nn_t *nn, double *output, double *A, double *Z, double *D, double *d, double *E, double *D_aux){
    int i, n_l;
    int *l_s;
    double loss;
    double T[600] = {0};

    n_l = nn->n_layers;
    l_s = nn->layers_size;

    int *size = &(l_s[1]);
    int *size_prev = &(l_s[0]);
    for (int u = 0; u < n_l - 1; u++){
        for (int v = 0; v < size[u] * size_prev[u]; v++){
            D_aux[threadId][u][v] = init_zero();
        }
        for (int v = 0; v < size[u]; v++){
            E[threadId][u][v] = init_zero();
        }
    }

    //loss = nn->loss(A[threadId][n_l - 1], output, l_s[n_l - 1]);
    //matrix_sum(E[threadId][n_l - 2], A[threadId][n_l - 1], A[threadId][n_l - 1], l_s[n_l - 1], 1);
    matrix_sub(E[threadId][n_l - 2], A[threadId][n_l - 1], output, l_s[n_l - 1], 1);
    matrix_mul_dot(E[threadId][n_l - 2], E[threadId][n_l - 2], Z[threadId][n_l - 1], l_s[n_l - 1], 1);  

    if (threadId == 0) {
        printf("OUT:\n");
        for (int i = 0; i < 10; i++) {
            printf(" %f", output[i]);
        }
        printf("\n");

        printf("Z[%d][%d]:\n", threadId, n_l - 1);
        for (int i = 0; i < 10; i++) {
            printf(" %f", Z[threadId][n_l - 1][i]);
        }
        printf("\n");

        printf("A[%d][%d]:\n", threadId, n_l - 1);
        for (int i = 0; i < 10; i++) {
            printf(" %f", A[threadId][n_l - 1][i]);
        }
        printf("\n");

        printf("E[%d][%d]:\n", threadId, n_l - 2);
        for (int i = 0; i < 10; i++) {
            printf(" %f", E[threadId][n_l - 2][i]);
        }
        printf("\n");
    }

    matrix_transpose_v2(A[threadId][n_l - 2], l_s[n_l - 2], 1, T); 
    matrix_mul(D_aux[threadId][n_l - 2], E[threadId][n_l - 2], T, l_s[n_l - 1], 1, 1, l_s[n_l - 2]);
    //matrix_free(T);

    matrix_sum(D[threadId][n_l - 2], D[threadId][n_l - 2], D_aux[threadId][n_l - 2], l_s[n_l - 1], l_s[n_l - 2]);
    matrix_sum(d[threadId][n_l - 2], d[threadId][n_l - 2], E[threadId][n_l - 2], l_s[n_l - 1], 1);

    for (i = n_l - 2; i > 0; i--) {
        matrix_transpose_v2(nn->WH[i], l_s[i + 1], l_s[i], T);
        matrix_mul(E[threadId][i - 1], T, E[threadId][i], l_s[i], l_s[i + 1], l_s[i + 1], 1);
        //matrix_free(T);

        matrix_mul_dot(E[threadId][i - 1], E[threadId][i - 1], Z[threadId][i], l_s[i], 1);

        matrix_mul(D_aux[threadId][i - 1], E[threadId][i - 1], A[threadId][i - 1], l_s[i], 1, 1, l_s[i - 1]);

        matrix_sum(D[threadId][i - 1], D[threadId][i - 1], D_aux[threadId][i - 1], l_s[i], l_s[i - 1]);
        matrix_sum(d[threadId][i - 1], d[threadId][i - 1], E[threadId][i - 1], l_s[i], 1);
    }
} */


/* __global__ void testZ(double ***array) {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    if (threadId == 1) {
        for (int k = 0; k < 10; k++) {
            printf("%f ", Z[1][0][k]);
        }
        printf("\n");
        for (int k = 0; k < 10; k++) {
            printf("%f ", Z[7641][0][k]);
        }
        printf("\n");
        for (int k = 0; k < 10; k++) {
            printf("%f ", Z[1782][0][k]);
        }
        printf("\n");
        for (int k = 0; k < 10; k++) {
            printf("%f ", Z[17082][0][k]);
        }
        printf("\n");
    }
} */

__global__ void testZ2(double *array) {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    int index;

    if (threadId == 1) {
        index = 15326 * (10 * 101) + 2 * 101;
        for (int k = 0; k < 10; k++) {
            printf("%f ", array[index + k]);
        }
        printf("\n");
    }
}

__global__ void test4Darray(double ****array) {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    if (threadId == 1) {
        for (int k = 0; k < 10; k++) {
            array[8402][7][1][k] = k * 1;
            array[7641][5][1][k] = k * 2;
            array[1782][1][1][k] = k * 3;
            array[17082][4][1][k] = k * 4;
        }


        for (int k = 0; k < 10; k++) {
            printf("%f ", array[8402][7][1][k]);
        }
        printf("\n");
        for (int k = 0; k < 10; k++) {
            printf("%f ", array[7641][5][1][k]);
        }
        printf("\n");
        for (int k = 0; k < 10; k++) {
            printf("%f ", array[1782][1][1][k]);
        }
        printf("\n");
        for (int k = 0; k < 10; k++) {
            printf("%f ", array[17082][4][1][k]);
        }
        printf("\n");
    }
}

template<typename T>
void array_to_device(T *&device, T *host, size_t size) {
    hipMalloc((void**)&device, size * sizeof(T));
    hipMemcpy(device, host, size * sizeof(T), hipMemcpyHostToDevice);
}

template<typename T>
void matrix_to_device(T *&device, T *host, size_t col_size, int *row_sizes, int layers) {
    hipMalloc((void***)(&device), col_size * sizeof(T));

    for (int i = 0; i < layers; i++) {
        double *array;
        array_to_device(array, host[i], row_sizes[i]);
        hipMemcpy(device + i, &array, sizeof(T*), hipMemcpyHostToDevice);
    }
}

template<typename T>
void matrix_to_device2(T *&device, T *host, size_t col_size, int *row_sizes, int *row_sizes_prev, int layers) {
    hipMalloc((void***)(&device), col_size * sizeof(T));

    for (int i = 0; i < layers; i++) {
        double *array;
        array_to_device(array, host[i], row_sizes[i] * row_sizes_prev[i]);
        hipMemcpy(device + i, &array, sizeof(T*), hipMemcpyHostToDevice);
    }
}

void train(nn_t *nn, ds_t *ds, int epochs, int size_batch, double lr) {
    /*------------------- Copy data to device -------------------*/
    ds_t *ds_d;
    double *inputs_d, *outputs_d, *max_d, *min_d, *mean_d, *std_d;

    hipMalloc((void**)&ds_d, sizeof(ds_t)); 
    hipMemcpy(ds_d, ds, sizeof(ds_t), hipMemcpyHostToDevice); 
    array_to_device(inputs_d, ds->inputs, ds->n_inputs * ds->n_samples);
    array_to_device(outputs_d, ds->outputs, ds->n_outputs * ds->n_samples);
    array_to_device(max_d, ds->max, ds->n_inputs);
    array_to_device(min_d, ds->min, ds->n_inputs);
    array_to_device(mean_d, ds->mean, ds->n_inputs);
    array_to_device(std_d, ds->std, ds->n_inputs);

    hipMemcpy(&(ds_d->inputs), &inputs_d, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(ds_d->outputs), &outputs_d, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(ds_d->max), &max_d, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(ds_d->min), &min_d, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(ds_d->mean), &mean_d, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(ds_d->std), &std_d, sizeof(double*), hipMemcpyHostToDevice);

    /*------------------- Copy NN to device -------------------*/
    nn_t *nn_d;
    int *layers_size_d;
    double **WH_d, **BH_d;

    hipMalloc((void**)&nn_d, sizeof(nn_t)); 
    hipMemcpy(nn_d, nn, sizeof(nn), hipMemcpyHostToDevice); 
    array_to_device(layers_size_d, nn->layers_size, nn->n_layers);
    matrix_to_device(BH_d, nn->BH, nn->layers_size[1] - 1, &(nn->layers_size[1]), nn->n_layers - 1);
    matrix_to_device2(WH_d, nn->WH, nn->layers_size[1] - 1, &(nn->layers_size[1]), &(nn->layers_size[0]), nn->n_layers - 1);

    hipMemcpy(&(nn_d->layers_size), &layers_size_d, sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(&(nn_d->BH), &BH_d, sizeof(double**), hipMemcpyHostToDevice);
    hipMemcpy(&(nn_d->WH), &WH_d, sizeof(double**), hipMemcpyHostToDevice);

    /*----- Initialize weights and gradients in devices -----*/
    //double **A, **Z, ***D, ***d, ***D_aux, ***E, *A_d, ****Z_d, ***D_d, ***d_d, ***D_aux_d, ***E_d, **temp;
    double *A, *Z, *D, *d, *D_aux, *E;
    int size = 0;
    int n_batches = ds->n_samples / size_batch;

    for (int i = 0; i < nn->n_layers; i++) {
        size += nn->layers_size[i];
    }

    // num batches * samples * layers * layer size
    hipMalloc(&A, n_batches * 10 * size * sizeof(double));
    hipMalloc(&Z, n_batches * 10 * size * sizeof(double));
    hipMalloc(&D, n_batches * 1 * size * sizeof(double));
    hipMalloc(&d, n_batches * 1 * size * sizeof(double));
    hipMalloc(&D_aux, n_batches * 1 * size * sizeof(double));
    hipMalloc(&E, n_batches * 1 * size * sizeof(double));
    hipMemset(A, 0, n_batches * 10 * size * sizeof(double));
    hipMemset(Z, 0, n_batches * 10 * size * sizeof(double));
    hipMemset(D, 0, n_batches * 1 * size * sizeof(double));
    hipMemset(d, 0, n_batches * 1 * size * sizeof(double));
    hipMemset(D_aux, 0, n_batches * 1 * size * sizeof(double));
    hipMemset(E, 0, n_batches * 1 * size * sizeof(double));

    printf("ON DEVICE\n");
    
    printf("%d\n", n_batches);
    int *order;
    order = (int*)malloc(ds->n_samples * sizeof(int));
    for(int i = 0; i < ds->n_samples; i++)
        order[i] = i;
    //shuffle(order, ds->n_samples);
    int *order_d;
    array_to_device(order_d, order, ds->n_samples);






    int block_col = 1;
    int reg = 625;
    int block_row = ceil(double(n_batches) / double(reg));
    dim3 thr_per_blk(1, reg);
    dim3 blk_in_grid(block_col, block_row);

    printf("Grid : {%d, %d} blocks. Blocks : {%d, %d} threads.\n", blk_in_grid.x, blk_in_grid.y, thr_per_blk.x, thr_per_blk.y);

    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    printf("START!\n");
    //forward_pass_kernel<<<blk_in_grid, thr_per_blk>>>(nn_d, ds_d, A_d, Z_d, n_batches, order_d);
    batch_forward_pass<<<blk_in_grid, thr_per_blk>>>(nn_d, ds_d, A, Z, 10, n_batches, order_d, size);
    hipDeviceSynchronize();
    testZ2<<<blk_in_grid, thr_per_blk>>>(Z);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kelnel run time: %f s\n", (milliseconds / 1000));
    
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
        exit(-1);
    } else {
        printf("All correct!\n");
    }



    //MEMORY LEAK ERROR !!!
/*     double **output_mat;
    hipMalloc((void***)(&output_mat), 199493 * sizeof(double));
    for (int i = 0; i < 199493; i++) {
        double *array;
        array_to_device(array, outputs_d, ds->n_outputs * ds->n_samples);
        hipMemcpy(output_mat + i, &array, sizeof(double*), hipMemcpyHostToDevice);
    } */
    
    //back_prop_kernel<<<blk_in_grid, thr_per_blk>>>(nn_d, &outputs_d[10 * ds->n_outputs], A_d, Z_d, D_d, d_d, E_d, D_aux_d);
    printf("BP PASSED!\n");

    hipDeviceSynchronize();
    printf("\n");
    //testZ2<<<blk_in_grid, thr_per_blk>>>(A_d, Z_d);
    hipDeviceSynchronize();
    printf("\n");
    //testZ<<<blk_in_grid, thr_per_blk>>>(d_d, D_d);

    hipFree(A);
    hipFree(Z);

    

    sleep(20000);
}














void print_nn(nn_t *nn){

    int i, j, k;
    
    printf("Layers (I/H/O)\n");

    for (i = 0; i < nn->n_layers; i++) {
        printf("%d ", nn->layers_size[i]);
    }
    printf("\n");
    
    printf("Hidden Biases\n ");

    for (i = 0; i < nn->n_layers - 1; i++) {
        for (j = 0; j < nn->layers_size[i + 1]; j++) {
            printf("%lf ", nn->BH[i][j]);
        }
        printf("\n");
    }

    printf("Hidden Weights\n ");
    
    for (i = 0; i < nn->n_layers - 1; i++) {
        for (j = 0; j < nn->layers_size[i + 1]; j++) {
            for(k = 0; k < nn->layers_size[i]; k++) {
                printf("%lf ", nn->WH[i][(j * nn->layers_size[i]) + k]);
            }
            printf("\n");
        }
    }

}

void import_nn(nn_t *nn, char *filename){

    int i, j, k;
    FILE *fd;

    if ((fd = fopen(filename,"r")) == NULL){
        perror("Error importing the model\n");
        exit(1);
    }
    
    fscanf(fd, "%d ", &n_layers);

    layers = (int*)malloc(n_layers * sizeof(int));

    for (i = 0; i < n_layers; i++) {
        fscanf(fd, "%d ", &(layers[i]));
    }

    init_nn(nn, n_layers, layers);
    
    for (i = 0; i < nn->n_layers - 1; i++) {
        for (j = 0; j < nn->layers_size[i + 1]; j++) {
            fscanf(fd, "%lf ", &(nn->BH[i][j]));
        }
    }

    for (i = 0; i < nn->n_layers - 1; i++) {
        for (j = 0; j < nn->layers_size[i + 1]; j++) {
            for(k = 0; k < nn->layers_size[i]; k++) {
                fscanf(fd, "%lf ", &(nn->WH[i][(j * nn->layers_size[i]) + k]));
            }
        }
    }
    fclose(fd);
}

void export_nn(nn_t *nn, char *filename){

    int i, j, k;
    FILE *fd;

    if ((fd = fopen(filename,"w")) == NULL){
        perror("Error exporting the model");
        exit(1);
    }
    
    fprintf(fd, "%d\n", nn->n_layers);

    for (i = 0; i < nn->n_layers; i++) {
        fprintf(fd, "%d ", nn->layers_size[i]);
    }
    fprintf(fd, "\n");
    
    for (i = 0; i < nn->n_layers - 1; i++) {
        for (j = 0; j < nn->layers_size[i + 1]; j++) {
            fprintf(fd, "%lf ", nn->BH[i][j]);
        }
        fprintf(fd, "\n");
    }

    for (i = 0; i < nn->n_layers - 1; i++) {
        for (j = 0; j < nn->layers_size[i + 1]; j++) {
            for(k = 0; k < nn->layers_size[i]; k++) {
                fprintf(fd, "%lf ", nn->WH[i][(j * nn->layers_size[i]) + k]);
            }
            fprintf(fd, "\n");
        }
    }
    fclose(fd);
}

