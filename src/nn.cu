#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h> 
#include <math.h>
#include <time.h>
#include "ds.h"
#include "nn.cuh"
#include "nn_aux.cuh"
#include "utils.h"
#include "matrix.cuh"
#include "test.h"
#include "train.h"
#include "globals.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <omp.h>

void init_nn(nn_t *nn, int n_layers, int *layers_size){
    int i;

    nn->n_layers = n_layers;
    nn->layers_size = layers_size;
    nn->init_weight_ptr = init_weight_rnd;
    nn->activation_ptr= (activation_ptr_t*)malloc((nn->n_layers - 1) * sizeof(activation_ptr_t));
    nn->dactivation_ptr= (activation_ptr_t*)malloc((nn->n_layers - 1) * sizeof(activation_ptr_t));
    for(i = 0; i < n_layers - 1; i++){
        nn->activation_ptr[i] = sigmoid;
        nn->dactivation_ptr[i] = dSigmoid;
    }
    nn->loss = mse;
    nn->BH = alloc_matrix_1v(n_layers - 1, &layers_size[1], nn->init_weight_ptr);
    nn->WH = alloc_matrix_2v(n_layers - 1, &layers_size[1], &layers_size[0], nn->init_weight_ptr);
}

__global__ void forward_pass_batch(nn_t *nn, ds_t *ds, double *A, double *Z, int batch_size, int batch_number, int *input_order) {
    int block_id = blockIdx.x + blockIdx.y * gridDim.x;
    int thread_id = block_id * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    unsigned int offset;
    int min_batch, i, matrix_size;

    if (thread_id < batch_number) {
        matrix_size = index_counter_1v(nn->layers_size, nn->n_layers);
        
        for (int sample = 0; sample < batch_size; sample++) {
            min_batch = thread_id * batch_size + sample;
            i = input_order[min_batch];

            offset = thread_id * (batch_size * matrix_size) + sample * matrix_size;
            forward_pass_kernel(nn, &ds->inputs[i * ds->n_inputs], &A[offset], &Z[offset]);
        }      
    }
}

__device__ void forward_pass_kernel(nn_t *nn, double *input, double *A, double *Z) {
    int offset_1, offset_2;

    for (int i = 0; i < nn->layers_size[0]; i++) {        
        A[i] = input[i];
    }
    
    for (int i = 1; i < nn->n_layers; i++) {
        offset_1 = index_counter_1v(nn->layers_size, i);
        offset_2 = index_counter_1v(nn->layers_size, i - 1);

        matrix_mul_add(&Z[offset_1], nn->WH[i - 1], &A[offset_2],  nn->layers_size[i], nn->layers_size[i - 1], nn->layers_size[i - 1], 1, nn->BH[i - 1]);  
        matrix_func(&A[offset_1], &Z[offset_1], nn->layers_size[i], 1, sigmoid);
        matrix_func(&Z[offset_1], &Z[offset_1], nn->layers_size[i], 1, dSigmoid);
    }
}

__global__ void back_prop_batch(nn_t *nn, ds_t *ds, double *A, double *Z, double *D, double *d, double *E, double *D_aux, int batch_size, int batch_number, int *input_order, double *loss) {
    int block_id = blockIdx.x + blockIdx.y * gridDim.x;
    int thread_id = block_id * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    unsigned int offset_1, offset_2, offset_3, offset_4;
    int min_batch, i;

    if (thread_id < batch_number) {
        for (int sample = 0; sample < batch_size; sample++) {
            min_batch = thread_id * batch_size + sample;
            i = input_order[min_batch];

            offset_1 = thread_id * (batch_size * 101) + sample * 101;
            offset_2 = thread_id * 2410;
            offset_3 = thread_id * 100;
            offset_4 = thread_id * 71;

            back_prop_kernel(nn, &ds->outputs[i * ds->n_outputs], &A[offset_1], &Z[offset_1], &D[offset_2], &d[offset_3], &E[offset_4], &D_aux[offset_2], &loss[thread_id]);
        }    

        loss[thread_id] = loss[thread_id] / batch_size;
    }
}

__device__ void back_prop_kernel(nn_t *nn, double *output, double *A, double *Z, double *D, double *d, double *E, double *D_aux, double *loss) {
    int i, n_l;
    int *l_s;
    double T[600] = {0};
    int offset_1, offset_2, offset_3, offset_4, offset_5, offset_6, offset_7, offset_8, offset_9;

    n_l = nn->n_layers;
    l_s = nn->layers_size;

    offset_1 = index_counter_1v(nn->layers_size, n_l - 1);                              // A Z d --> n_l - 1
    offset_2 = index_counter_1v(nn->layers_size, n_l - 2);                              // A Z d --> n_l - 2
    offset_3 = index_counter_2v(&(nn->layers_size[1]), &(nn->layers_size[0]), n_l - 2); // D D_aux --> n_l - 2
    offset_4 = index_counter_1v(&(nn->layers_size[1]), n_l - 2);                        // E --> n_l - 2

    *loss += mse(&A[offset_1], output, l_s[n_l - 1]);

    matrix_sub(&E[offset_4], &A[offset_1], output, l_s[n_l - 1], 1);
    matrix_mul_dot(&E[offset_4], &E[offset_4], &Z[offset_1], l_s[n_l - 1], 1);  

    matrix_transpose_v2(&A[offset_2], l_s[n_l - 2], 1, T); 
    matrix_mul(&D_aux[offset_3], &E[offset_4], T, l_s[n_l - 1], 1, 1, l_s[n_l - 2]);

    matrix_sum(&D[offset_3], &D[offset_3], &D_aux[offset_3], l_s[n_l - 1], l_s[n_l - 2]);
    matrix_sum(&d[offset_2], &d[offset_2], &E[offset_4], l_s[n_l - 1], 1);

    for (i = n_l - 2; i > 0; i--) {
        offset_5 = index_counter_1v(&(nn->layers_size[1]), i - 1);                          // E --> i - 1
        offset_6 = index_counter_1v(&(nn->layers_size[1]), i);                              // E --> i
        offset_7 = index_counter_1v(nn->layers_size, i);                                    // A Z d --> i - 1
        offset_8 = index_counter_1v(nn->layers_size, i);                                    // A Z d --> i
        offset_9 = index_counter_2v(&(nn->layers_size[1]), &(nn->layers_size[0]), i - 1);   // D D_aux --> i - 1

        matrix_transpose_v2(nn->WH[i], l_s[i + 1], l_s[i], T);
        matrix_mul(&E[offset_5], T, &E[offset_6], l_s[i], l_s[i + 1], l_s[i + 1], 1);

        matrix_mul_dot(&E[offset_5], &E[offset_5], &Z[offset_8], l_s[i], 1);

        matrix_mul(&D_aux[offset_9], &E[offset_5], &A[offset_7], l_s[i], 1, 1, l_s[i - 1]);

        matrix_sum(&D[offset_9], &D[offset_9], &D_aux[offset_9], l_s[i], l_s[i - 1]);
        matrix_sum(&d[offset_7], &d[offset_7], &E[offset_5], l_s[i], 1);
    }
}

__global__ void gradient_average(double *gradient_batch, double *gradient_avg, int size, int batch_number) {
    int block_id = blockIdx.x + blockIdx.y * gridDim.x;
    int thread_id = block_id * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    int offset;

    if (thread_id < size) {
        for (int j = 0; j < batch_number; j++) {
            offset = (j * size) + thread_id;
            gradient_avg[thread_id] += gradient_batch[offset];
        }
    }
}

__global__ void update(nn_t *nn, double *D, double *d, double lr, int batch_size) {
    int block_id = blockIdx.x + blockIdx.y * gridDim.x;
    int thread_id = block_id * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    int i, offset;

    if (thread_id == 0) {
        for(i = 0; i < nn->n_layers - 1; i++){
            offset = index_counter_2v(&(nn->layers_size[1]), &(nn->layers_size[0]), i); 

            matrix_mul_cnt(&D[offset], nn->layers_size[i + 1], nn->layers_size[i],  lr * (1.0 / batch_size));
            matrix_mul_cnt(&d[offset], nn->layers_size[i + 1], 1,  lr * (1.0 / batch_size));
            matrix_sub(nn->WH[i], nn->WH[i], &D[offset],  nn->layers_size[i + 1], nn->layers_size[i]);
            matrix_sub(nn->BH[i], nn->BH[i], &d[offset],  nn->layers_size[i + 1], 1);
        }
    } 
}

__global__ void test_train(ds_t *ds) {
    int block_id = blockIdx.x + blockIdx.y * gridDim.x;
    int thread_id = block_id * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    int i;

    if (thread_id == 0) {
        printf("HELLOW FROM %d\n", ds->n_samples);

        for (int i = 0; i < 10; i++) {
            printf("%d ", ds->inputs[i + 2000]);
        }

    }
}

void train(nn_t *nn, ds_t *ds, int epochs, int size_batch, double lr) {
    /*------------------- Multi GPU -------------------*/
    int CUDA_device_count, CPU_thread_count, active_devices;
    hipGetDeviceCount(&CUDA_device_count);
    CPU_thread_count = omp_get_max_threads();
    active_devices = (CUDA_device_count < CPU_thread_count) ? CUDA_device_count : CPU_thread_count;

    if(verbose)
        printf("CUDA devices: %d    CPU threads: %d\n", CUDA_device_count, CPU_thread_count);

    ds_t** ds_d = (ds_t**)malloc(active_devices * sizeof(ds_t*));
    nn_t** nn_d = (nn_t**)malloc(active_devices * sizeof(nn_t*));

    /*------------------- Copy data and NN to each device  -------------------*/
    #pragma omp parallel for shared(ds_d, nn_d) num_threads(active_devices)
    for (int device = 0; device < active_devices; device++) {
        hipSetDevice(device);

        /*------------------- Copy data to device -------------------*/
        double *inputs_d, *outputs_d, *max_d, *min_d, *mean_d, *std_d;

        hipMalloc((void**)&(ds_d[device]), sizeof(ds_t)); 
        hipMemcpy(ds_d[device], ds, sizeof(ds_t), hipMemcpyHostToDevice); 

        array_to_device(inputs_d, ds->inputs, ds->n_inputs * ds->n_samples);
        array_to_device(outputs_d, ds->outputs, ds->n_outputs * ds->n_samples);
        array_to_device(max_d, ds->max, ds->n_inputs);
        array_to_device(min_d, ds->min, ds->n_inputs);
        array_to_device(mean_d, ds->mean, ds->n_inputs);
        array_to_device(std_d, ds->std, ds->n_inputs);

        hipMemcpy(&(ds_d[device]->inputs), &inputs_d, sizeof(double*), hipMemcpyHostToDevice);
        hipMemcpy(&(ds_d[device]->outputs), &outputs_d, sizeof(double*), hipMemcpyHostToDevice);
        hipMemcpy(&(ds_d[device]->max), &max_d, sizeof(double*), hipMemcpyHostToDevice);
        hipMemcpy(&(ds_d[device]->min), &min_d, sizeof(double*), hipMemcpyHostToDevice);
        hipMemcpy(&(ds_d[device]->mean), &mean_d, sizeof(double*), hipMemcpyHostToDevice);
        hipMemcpy(&(ds_d[device]->std), &std_d, sizeof(double*), hipMemcpyHostToDevice);

        /*------------------- Copy NN to device -------------------*/
        nn_t *nn_thread;
        int *layers_size_d;
        double **WH_d, **BH_d;
        
        hipMalloc((void**)&(nn_d[device]), sizeof(nn_t)); 
        hipMemcpy(nn_d[device], nn, sizeof(nn), hipMemcpyHostToDevice); 

        array_to_device(layers_size_d, nn->layers_size, nn->n_layers);
        matrix_to_device_v1(BH_d, nn->BH, nn->layers_size[1] - 1, &(nn->layers_size[1]), nn->n_layers - 1);
        matrix_to_device_v2(WH_d, nn->WH, nn->layers_size[1] - 1, &(nn->layers_size[1]), &(nn->layers_size[0]), nn->n_layers - 1);

        hipMemcpy(&(nn_d[device]->layers_size), &layers_size_d, sizeof(int*), hipMemcpyHostToDevice);
        hipMemcpy(&(nn_d[device]->BH), &BH_d, sizeof(double**), hipMemcpyHostToDevice);
        hipMemcpy(&(nn_d[device]->WH), &WH_d, sizeof(double**), hipMemcpyHostToDevice);
    }

    //test_train<<<blk_in_grid_1, thr_per_blk_1>>>(ds_d[0]);

    /*----- Initialize weights and gradients in devices -----*/
    double *A, *Z, *D, *d, *D_aux, *E, *loss, *avg_D, *avg_d, *loss_h, loss_value;
    int size_1, size_2, size_3, size_4, n_batches, *order, *order_d;
    hipEvent_t start, stop;
    float milliseconds = 0;

    n_batches = ds->n_samples / size_batch;
    size_1 = index_counter_1v(nn->layers_size, nn->n_layers);
    size_2 = index_counter_2v(&(nn->layers_size[1]), &(nn->layers_size[0]), nn->n_layers- 1);
    size_3 = index_counter_1v(nn->layers_size, nn->n_layers - 1);
    size_4 = index_counter_1v(&(nn->layers_size[1]), nn->n_layers - 1);

    hipMalloc(&A, n_batches * size_batch * size_1 * sizeof(double));   // 4D tensor
    hipMalloc(&Z, n_batches * size_batch * size_1 * sizeof(double));   // 4D tensor
    hipMalloc(&D, n_batches * size_2 * sizeof(double));                // 3D tensor
    hipMalloc(&d, n_batches * size_3 * sizeof(double));                // 3D tensor
    hipMalloc(&D_aux, n_batches * size_2 * sizeof(double));            // 3D tensor
    hipMalloc(&E, n_batches * size_4 * sizeof(double));                // 3D tensor
    hipMalloc(&avg_D, size_2 * sizeof(double));                        // 2D tensor
    hipMalloc(&avg_d, size_3 * sizeof(double));                        // 2D tensor
    hipMalloc(&loss, n_batches * sizeof(double));                      // 1D tensor

    hipMemset(A, 0, n_batches * size_batch * size_1 * sizeof(double));
    hipMemset(Z, 0, n_batches * size_batch * size_1 * sizeof(double));
    hipMemset(D, 0, n_batches * size_2 * sizeof(double));
    hipMemset(d, 0, n_batches * size_3 * sizeof(double));
    hipMemset(D_aux, 0, n_batches * size_2 * sizeof(double));
    hipMemset(E, 0, n_batches * size_4 * sizeof(double));
    hipMemset(loss, 0, n_batches * sizeof(double));
    hipMemset(avg_D, 0, size_2 * sizeof(double));
    hipMemset(avg_d, 0, size_3 * sizeof(double));

    order = (int*)malloc(ds->n_samples * sizeof(int));
    loss_h = (double*)malloc(n_batches * sizeof(double));

    for (int i = 0; i < ds->n_samples; i++)
        order[i] = i;

    // CUDA kernel launch config
    int thr_col = 32;
    int thr_row = 20;
    int block_col = ceil(double(n_batches) / double(thr_col));
    int block_row = ceil(double(n_batches) / double(thr_row));
    dim3 thr_per_blk(thr_col, thr_row);
    dim3 blk_in_grid(block_col, block_row);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    /*----- Train process -----*/
    for (int n = 0; n < epochs; n++) {

        if(verbose)
            printf("Epoch %d/%d \n", n, epochs);

        hipMemset(loss, 0, n_batches * sizeof(double));
        shuffle(order, ds->n_samples);
        array_to_device(order_d, order, ds->n_samples);
        
        hipEventRecord(start);

        //Start parallel section
        forward_pass_batch<<<blk_in_grid, thr_per_blk>>>(nn_d[0], ds_d[0], A, Z, size_batch, n_batches, order_d);
        back_prop_batch<<<blk_in_grid, thr_per_blk>>>(nn_d[0], ds_d[0], A, Z, D, d, E, D_aux, size_batch, n_batches, order_d, loss);
        gradient_average<<<blk_in_grid, thr_per_blk>>>(D, avg_D, size_2, n_batches);
        gradient_average<<<blk_in_grid, thr_per_blk>>>(d, avg_d, size_3, n_batches);
        //End parallel section

        //Average gradients across devices

        //Start parallel section
        update<<<blk_in_grid, thr_per_blk>>>(nn_d[0], avg_D, avg_d, lr, n_batches);
        //End parallel section

        hipMemset(D, 0, n_batches * 1 * size_2 * sizeof(double));
        hipMemset(d, 0, n_batches * 1 * size_3 * sizeof(double));

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);

        hipMemcpy(loss_h, loss, n_batches * sizeof(double), hipMemcpyDeviceToHost);
        loss_value = array_sum(loss_h, n_batches) / n_batches;

        if(verbose)
            printf(" time: %f s - loss: %f\n", (milliseconds / 1000), loss_value);
        
        hipFree(order_d);
    }

    hipFree(A);
    hipFree(Z);
    hipFree(D);
    hipFree(d);
    hipFree(D_aux);
    hipFree(E);
    hipFree(loss);
    hipFree(avg_D);
    hipFree(avg_d);

}


void print_nn(nn_t *nn){

    int i, j, k;
    
    printf("Layers (I/H/O)\n");

    for (i = 0; i < nn->n_layers; i++) {
        printf("%d ", nn->layers_size[i]);
    }
    printf("\n");
    
    printf("Hidden Biases\n ");

    for (i = 0; i < nn->n_layers - 1; i++) {
        for (j = 0; j < nn->layers_size[i + 1]; j++) {
            printf("%lf ", nn->BH[i][j]);
        }
        printf("\n");
    }

    printf("Hidden Weights\n ");
    
    for (i = 0; i < nn->n_layers - 1; i++) {
        for (j = 0; j < nn->layers_size[i + 1]; j++) {
            for(k = 0; k < nn->layers_size[i]; k++) {
                printf("%lf ", nn->WH[i][(j * nn->layers_size[i]) + k]);
            }
            printf("\n");
        }
    }

}

void import_nn(nn_t *nn, char *filename){

    int i, j, k;
    FILE *fd;

    if ((fd = fopen(filename,"r")) == NULL){
        perror("Error importing the model\n");
        exit(1);
    }
    
    fscanf(fd, "%d ", &n_layers);

    layers = (int*)malloc(n_layers * sizeof(int));

    for (i = 0; i < n_layers; i++) {
        fscanf(fd, "%d ", &(layers[i]));
    }

    init_nn(nn, n_layers, layers);
    
    for (i = 0; i < nn->n_layers - 1; i++) {
        for (j = 0; j < nn->layers_size[i + 1]; j++) {
            fscanf(fd, "%lf ", &(nn->BH[i][j]));
        }
    }

    for (i = 0; i < nn->n_layers - 1; i++) {
        for (j = 0; j < nn->layers_size[i + 1]; j++) {
            for(k = 0; k < nn->layers_size[i]; k++) {
                fscanf(fd, "%lf ", &(nn->WH[i][(j * nn->layers_size[i]) + k]));
            }
        }
    }
    fclose(fd);
}

void export_nn(nn_t *nn, char *filename){

    int i, j, k;
    FILE *fd;

    if ((fd = fopen(filename,"w")) == NULL){
        perror("Error exporting the model");
        exit(1);
    }
    
    fprintf(fd, "%d\n", nn->n_layers);

    for (i = 0; i < nn->n_layers; i++) {
        fprintf(fd, "%d ", nn->layers_size[i]);
    }
    fprintf(fd, "\n");
    
    for (i = 0; i < nn->n_layers - 1; i++) {
        for (j = 0; j < nn->layers_size[i + 1]; j++) {
            fprintf(fd, "%lf ", nn->BH[i][j]);
        }
        fprintf(fd, "\n");
    }

    for (i = 0; i < nn->n_layers - 1; i++) {
        for (j = 0; j < nn->layers_size[i + 1]; j++) {
            for(k = 0; k < nn->layers_size[i]; k++) {
                fprintf(fd, "%lf ", nn->WH[i][(j * nn->layers_size[i]) + k]);
            }
            fprintf(fd, "\n");
        }
    }
    fclose(fd);
}

